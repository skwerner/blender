
#include <hip/hip_runtime.h>
/*
 * Copyright 2011-2013 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* CUDA kernel entry points */

#ifdef __CUDA_ARCH__

#  include "kernel/device/cuda/compat.h"
#  include "kernel/device/cuda/config.h"
#  include "kernel/device/cuda/globals.h"
#  include "kernel/device/cuda/image.h"
#  include "kernel/device/cuda/parallel_active_index.h"
#  include "kernel/device/cuda/parallel_prefix_sum.h"
#  include "kernel/device/cuda/parallel_sorted_index.h"

#  include "kernel/integrator/integrator_state.h"
#  include "kernel/integrator/integrator_state_flow.h"
#  include "kernel/integrator/integrator_state_util.h"

#  include "kernel/integrator/integrator_init_from_camera.h"
#  include "kernel/integrator/integrator_intersect_closest.h"
#  include "kernel/integrator/integrator_intersect_shadow.h"
#  include "kernel/integrator/integrator_intersect_subsurface.h"
#  include "kernel/integrator/integrator_shade_background.h"
#  include "kernel/integrator/integrator_shade_light.h"
#  include "kernel/integrator/integrator_shade_shadow.h"
#  include "kernel/integrator/integrator_shade_surface.h"
#  include "kernel/integrator/integrator_shade_volume.h"

#  include "kernel/kernel_adaptive_sampling.h"
#  include "kernel/kernel_bake.h"
#  include "kernel/kernel_film.h"
#  include "kernel/kernel_work_stealing.h"

/* TODO: move cryptomatte post sorting to its own kernel. */
#  if 0
/* kernels */
extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS, CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_path_trace(KernelWorkTile *tile, uint work_size)
{
  int work_index = ccl_global_id(0);
  bool thread_is_active = work_index < work_size;
  uint x, y, sample;
  KernelGlobals kg;
  if(thread_is_active) {
    get_work_pixel(tile, work_index, &x, &y, &sample);

    kernel_path_trace(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
  }

  if(kernel_data.film.cryptomatte_passes) {
    __syncthreads();
    if(thread_is_active) {
      kernel_cryptomatte_post(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
    }
  }
}
#  endif

/* --------------------------------------------------------------------
 * Integrator.
 */

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_reset(int num_states, int num_keys)
{
  const int path_index = ccl_global_id(0);

  if (path_index < num_states) {
    INTEGRATOR_STATE_WRITE(path, queued_kernel) = 0;
    INTEGRATOR_STATE_WRITE(shadow_path, queued_kernel) = 0;
  }

  if (path_index < num_keys) {
    kernel_integrator_state.sort_key_counter[path_index] = 0;
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_init_from_camera(const int *path_index_array,
                                            KernelWorkTile *tiles,
                                            const int num_tiles,
                                            float *render_buffer,
                                            const int max_tile_work_size)
{
  const int work_index = ccl_global_id(0);

  if (work_index >= max_tile_work_size * num_tiles) {
    return;
  }

  const int tile_index = work_index / max_tile_work_size;
  const int tile_work_index = work_index - tile_index * max_tile_work_size;

  const KernelWorkTile *tile = &tiles[tile_index];

  if (tile_work_index >= tile->work_size) {
    return;
  }

  const int path_index = (path_index_array) ?
                             path_index_array[tile->path_index_offset + tile_work_index] :
                             tile->path_index_offset + tile_work_index;

  uint x, y, sample;
  get_work_pixel(tile, tile_work_index, &x, &y, &sample);

  integrator_init_from_camera(nullptr, path_index, tile, render_buffer, x, y, sample);
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_intersect_closest(const int *path_index_array, const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_intersect_closest(NULL, path_index);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_intersect_shadow(const int *path_index_array, const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_intersect_shadow(NULL, path_index);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_intersect_subsurface(const int *path_index_array, const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_intersect_subsurface(NULL, path_index);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_shade_background(const int *path_index_array,
                                            float *render_buffer,
                                            const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_shade_background(NULL, path_index, render_buffer);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_shade_light(const int *path_index_array,
                                       float *render_buffer,
                                       const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_shade_light(NULL, path_index, render_buffer);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_shade_shadow(const int *path_index_array,
                                        float *render_buffer,
                                        const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_shade_shadow(NULL, path_index, render_buffer);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_shade_surface(const int *path_index_array,
                                         float *render_buffer,
                                         const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_shade_surface(NULL, path_index, render_buffer);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_integrator_shade_volume(const int *path_index_array,
                                        float *render_buffer,
                                        const int work_size)
{
  const int global_index = ccl_global_id(0);

  if (global_index < work_size) {
    const int path_index = (path_index_array) ? path_index_array[global_index] : global_index;
    integrator_shade_volume(NULL, path_index, render_buffer);
  }
}

extern "C" __global__ void __launch_bounds__(CUDA_PARALLEL_ACTIVE_INDEX_DEFAULT_BLOCK_SIZE)
    kernel_cuda_integrator_queued_paths_array(int num_states,
                                              int *indices,
                                              int *num_indices,
                                              int kernel)
{
  cuda_parallel_active_index_array<CUDA_PARALLEL_ACTIVE_INDEX_DEFAULT_BLOCK_SIZE>(
      num_states, indices, num_indices, [kernel](const int path_index) {
        return (INTEGRATOR_STATE(path, queued_kernel) == kernel);
      });
}

extern "C" __global__ void __launch_bounds__(CUDA_PARALLEL_ACTIVE_INDEX_DEFAULT_BLOCK_SIZE)
    kernel_cuda_integrator_queued_shadow_paths_array(int num_states,
                                                     int *indices,
                                                     int *num_indices,
                                                     int kernel)
{
  cuda_parallel_active_index_array<CUDA_PARALLEL_ACTIVE_INDEX_DEFAULT_BLOCK_SIZE>(
      num_states, indices, num_indices, [kernel](const int path_index) {
        return (INTEGRATOR_STATE(shadow_path, queued_kernel) == kernel);
      });
}

extern "C" __global__ void __launch_bounds__(CUDA_PARALLEL_ACTIVE_INDEX_DEFAULT_BLOCK_SIZE)
    kernel_cuda_integrator_active_paths_array(int num_states,
                                              int *indices,
                                              int *num_indices,
                                              int unused_kernel)
{
  cuda_parallel_active_index_array<CUDA_PARALLEL_ACTIVE_INDEX_DEFAULT_BLOCK_SIZE>(
      num_states, indices, num_indices, [](const int path_index) {
        return (INTEGRATOR_STATE(path, queued_kernel) != 0) ||
               (INTEGRATOR_STATE(shadow_path, queued_kernel) != 0);
      });
}

extern "C" __global__ void __launch_bounds__(CUDA_PARALLEL_ACTIVE_INDEX_DEFAULT_BLOCK_SIZE)
    kernel_cuda_integrator_terminated_paths_array(int num_states,
                                                  int *indices,
                                                  int *num_indices,
                                                  int unused_kernel)
{
  cuda_parallel_active_index_array<CUDA_PARALLEL_ACTIVE_INDEX_DEFAULT_BLOCK_SIZE>(
      num_states, indices, num_indices, [](const int path_index) {
        if (kernel_data.integrator.has_shadow_catcher) {
          /* NOTE: The kernel invocation limits number of states checked, ensuring that only
           * non-shadow-catcher states are checked here. */

          /* Only allow termination of both complementary states did finish their job. */
          if (INTEGRATOR_SHADOW_CATCHER_STATE(path, queued_kernel) != 0 ||
              INTEGRATOR_SHADOW_CATCHER_STATE(shadow_path, queued_kernel) != 0) {
            return false;
          }
        }
        return (INTEGRATOR_STATE(path, queued_kernel) == 0) &&
               (INTEGRATOR_STATE(shadow_path, queued_kernel) == 0);
      });
}

extern "C" __global__ void __launch_bounds__(CUDA_PARALLEL_SORTED_INDEX_DEFAULT_BLOCK_SIZE)
    kernel_cuda_integrator_sorted_paths_array(
        int num_states, int *indices, int *num_indices, int *key_prefix_sum, int kernel)
{
  cuda_parallel_sorted_index_array<CUDA_PARALLEL_SORTED_INDEX_DEFAULT_BLOCK_SIZE>(
      num_states, indices, num_indices, key_prefix_sum, [kernel](const int path_index) {
        return (INTEGRATOR_STATE(path, queued_kernel) == kernel) ?
                   INTEGRATOR_STATE(path, shader_sort_key) :
                   CUDA_PARALLEL_SORTED_INDEX_INACTIVE_KEY;
      });
}

extern "C" __global__ void __launch_bounds__(CUDA_PARALLEL_PREFIX_SUM_DEFAULT_BLOCK_SIZE)
    kernel_cuda_prefix_sum(int *values, int num_values)
{
  cuda_parallel_prefix_sum<CUDA_PARALLEL_PREFIX_SUM_DEFAULT_BLOCK_SIZE>(values, num_values);
}

/* --------------------------------------------------------------------
 * Adaptive sampling.
 */

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_adaptive_sampling_convergence_check(float *render_buffer,
                                                    int sx,
                                                    int sy,
                                                    int sw,
                                                    int sh,
                                                    float threshold,
                                                    bool reset,
                                                    int offset,
                                                    int stride,
                                                    uint *num_active_pixels)
{
  const int work_index = ccl_global_id(0);
  const int y = work_index / sw;
  const int x = work_index - y * sw;

  bool converged = true;

  if (x < sw && y < sh) {
    converged = kernel_adaptive_sampling_convergence_check(
        nullptr, render_buffer, sx + x, sy + y, threshold, reset, offset, stride);
  }

  /* NOTE: All threads specified in the mask must execute the intrinsic. */
  const uint num_active_pixels_mask = __ballot_sync(0xffffffff, !converged);
  if (threadIdx.x == 0) {
    atomic_fetch_and_add_uint32(num_active_pixels, __popc(num_active_pixels_mask));
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_adaptive_sampling_filter_x(
        float *render_buffer, int sx, int sy, int sw, int sh, int offset, int stride)
{
  const int y = ccl_global_id(0);

  if (y < sh) {
    kernel_adaptive_sampling_filter_x(NULL, render_buffer, sy + y, sx, sw, offset, stride);
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_adaptive_sampling_filter_y(
        float *render_buffer, int sx, int sy, int sw, int sh, int offset, int stride)
{
  const int x = ccl_global_id(0);

  if (x < sw) {
    kernel_adaptive_sampling_filter_y(NULL, render_buffer, sx + x, sy, sh, offset, stride);
  }
}

/* --------------------------------------------------------------------
 * Film.
 */

/* Convert to Display Buffer */

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_convert_to_half_float(uchar4 *rgba,
                                      float *render_buffer,
                                      float sample_scale,
                                      int sx,
                                      int sy,
                                      int sw,
                                      int sh,
                                      int offset,
                                      int stride)
{
  const int work_index = ccl_global_id(0);
  const int y = work_index / sw;
  const int x = work_index - y * sw;

  if (x < sw && y < sh) {
    kernel_film_convert_to_half_float(
        NULL, rgba, render_buffer, sample_scale, sx + x, sy + y, offset, stride);
  }
}

/* --------------------------------------------------------------------
 * Shader evaluaiton.
 */

/* Displacement */

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_shader_eval_displace(KernelShaderEvalInput *input,
                                     float4 *output,
                                     const int offset,
                                     const int work_size)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < work_size) {
    kernel_displace_evaluate(NULL, input, output, offset + i);
  }
}

/* Background Shader Evaluation */

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_shader_eval_background(KernelShaderEvalInput *input,
                                       float4 *output,
                                       const int offset,
                                       const int work_size)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < work_size) {
    kernel_background_evaluate(NULL, input, output, offset + i);
  }
}

/* --------------------------------------------------------------------
 * Baking.
 */

#  ifdef __BAKING__
extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_bake(KernelWorkTile *tile, uint work_size)
{
  /* TODO */
#    if 0
  int work_index = ccl_global_id(0);

  if (work_index < work_size) {
    uint x, y, sample;
    get_work_pixel(tile, work_index, &x, &y, &sample);

    KernelGlobals kg;
    kernel_bake_evaluate(&kg, tile->buffer, sample, x, y, tile->offset, tile->stride);
  }
#    endif
}
#  endif

/* --------------------------------------------------------------------
 * Denoising.
 */

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_filter_convert_to_rgb(float *rgb,
                                      const float *render_buffer,
                                      int sx,
                                      int sy,
                                      int sw,
                                      int sh,
                                      int offset,
                                      int stride,
                                      int pass_stride,
                                      int3 pass_offset,
                                      int num_inputs,
                                      int num_samples,
                                      int pass_sample_count)
{
  const int work_index = ccl_global_id(0);
  const int y = work_index / sw;
  const int x = work_index - y * sw;

  if (x >= sw || y >= sh) {
    return;
  }

  const int render_pixel_index = offset + (x + sx) + (y + sy) * stride;
  const float *buffer = render_buffer + (uint64_t)render_pixel_index * pass_stride;

  float pixel_scale;
  if (pass_sample_count == PASS_UNUSED) {
    pixel_scale = 1.0f / num_samples;
  }
  else {
    pixel_scale = 1.0f / __float_as_uint(buffer[pass_sample_count]);
  }

  if (num_inputs > 0) {
    const float *in = buffer + pass_offset.x;
    float *out = rgb + (x + y * sw) * 3;
    out[0] = clamp(in[0] * pixel_scale, 0.0f, 10000.0f);
    out[1] = clamp(in[1] * pixel_scale, 0.0f, 10000.0f);
    out[2] = clamp(in[2] * pixel_scale, 0.0f, 10000.0f);
  }

  if (num_inputs > 1) {
    const float *in = buffer + pass_offset.y;
    float *out = rgb + (x + y * sw) * 3 + (sw * sh) * 3;
    out[0] = in[0] * pixel_scale;
    out[1] = in[1] * pixel_scale;
    out[2] = in[2] * pixel_scale;
  }

  if (num_inputs > 2) {
    const float *in = buffer + pass_offset.z;
    float *out = rgb + (x + y * sw) * 3 + (sw * sh * 2) * 3;
    out[0] = in[0] * pixel_scale;
    out[1] = in[1] * pixel_scale;
    out[2] = in[2] * pixel_scale;
  }
}

extern "C" __global__ void CUDA_LAUNCH_BOUNDS(CUDA_KERNEL_BLOCK_NUM_THREADS,
                                              CUDA_KERNEL_MAX_REGISTERS)
    kernel_cuda_filter_convert_from_rgb(const float *rgb,
                                        float *render_buffer,
                                        int sx,
                                        int sy,
                                        int sw,
                                        int sh,
                                        int offset,
                                        int stride,
                                        int pass_stride,
                                        int num_samples,
                                        int pass_sample_count)
{
  const int work_index = ccl_global_id(0);
  const int y = work_index / sw;
  const int x = work_index - y * sw;

  if (x >= sw || y >= sh) {
    return;
  }

  const float *in = rgb + (x + y * sw) * 3;

  const int render_pixel_index = offset + (x + sx) + (y + sy) * stride;
  float *buffer = render_buffer + (uint64_t)render_pixel_index * pass_stride;

  float pixel_scale;
  if (pass_sample_count == PASS_UNUSED) {
    pixel_scale = num_samples;
  }
  else {
    pixel_scale = __float_as_uint(buffer[pass_sample_count]);
  }

  buffer[0] = in[0] * pixel_scale;
  buffer[1] = in[1] * pixel_scale;
  buffer[2] = in[2] * pixel_scale;
}

#endif
